#include "hip/hip_runtime.h"
// #include <__clang_cuda_builtin_vars.h>
// #include <__clang_cuda_runtime_wrapper.h>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define INCORRECT_NUM_CHANNELS -3
#include "stb/stb_image.h"
#include "stb/stb_image_write.h"

#include <cmath>

typedef struct Image {
  uint8_t *img_arr;
  int height;
  int width;
  int channels;
  int img_size;
} Image;

__global__ void greyScaling_kernel(uint8_t *ans, uint8_t *input,
                                   int threads_needed, int channels) {
  //

  int thread_id = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (thread_id < threads_needed) {
    int start_id = thread_id * channels;
    int r = input[start_id];
    int g = input[start_id + 1];
    int b = input[start_id + 2];

    uint8_t grey_pixel = (uint8_t)(0.2 * r + 0.2 * g + 0.2 * b);

    ans[start_id] = grey_pixel;
    ans[start_id + 1] = grey_pixel;
    ans[start_id + 2] = grey_pixel;
  } else {
    printf("why am i here ? just to suffer!!\n");
  }
}

int main() {
  //
  Image loaded_img;

  // last arg to force RGB image
  loaded_img.img_arr = stbi_load("some_image.png", &loaded_img.width,
                                 &loaded_img.height, &loaded_img.channels, 3);

  loaded_img.img_size =
      loaded_img.channels * loaded_img.height * loaded_img.width;

  printf("Img size : %d\n", loaded_img.img_size);

  // run settings
  // we will use one thread to evaluate one pixel i.e 3 elements of img_arr
  int threads_needed = loaded_img.img_size / loaded_img.channels;
  printf("Threads needed to process the image : %d\n", threads_needed);

  // 512 / 32 = 16 warps ... is optimal for new cards!
  int threads_per_block = 512;
  int blocks = threads_needed / threads_per_block;

  // allocate memory and send it to gpu
  uint8_t *result_d, *input_d;
  uint8_t *result_h = (uint8_t *)malloc(loaded_img.img_size);

  hipMalloc((void **)&result_d, loaded_img.img_size);
  hipMalloc((void **)&input_d, loaded_img.img_size);

  hipMemcpy(input_d, loaded_img.img_arr, loaded_img.img_size,
             hipMemcpyHostToDevice);

  greyScaling_kernel<<<blocks, threads_per_block>>>(
      result_d, input_d, threads_needed, loaded_img.channels);

  // take back answer to the host
  hipMemcpy(result_h, result_d, loaded_img.img_size, hipMemcpyDeviceToHost);
  hipFree(result_d);
  hipFree(result_h);

  // write the image!
  stbi_write_png("greyscaled_by_gpu.png", loaded_img.width, loaded_img.height,
                 loaded_img.channels, result_h,
                 loaded_img.width * loaded_img.channels);

  free(result_h);
  stbi_image_free(loaded_img.img_arr);
  return 0;
}
